/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "internal/device.h"
#include "kernels/concat_kernels.h"
#include <cassert>

namespace FlexFlow::Kernels::Concat {

void calc_blk_size(size_t &num_blocks,
                   size_t &blk_size,
                   ArrayShape const &shape,
                   ff_dim_t axis) {
  legion_dim_t legion_axis = legion_dim_from_ff_dim(axis, shape.num_dims());
  assert(legion_axis.value < shape.num_dims());
  if (legion_axis.value == 0_n) {
    legion_axis.value = 1_n;
  }
  blk_size = shape.sub_shape(legion_dim_t{0_n}, legion_axis)
                 .num_elements()
                 .unwrap_nonnegative();
  num_blocks = shape.sub_shape(legion_axis, std::nullopt)
                   .num_elements()
                   .unwrap_nonnegative();
}

void forward_kernel(hipStream_t stream,
                    GenericTensorAccessorW const &output,
                    std::vector<GenericTensorAccessorR> const &inputs,
                    ff_dim_t axis) {
  assert(inputs.size() <= MAX_NUM_INPUTS);
  size_t num_blocks = 1, output_blk_size = 1;
  calc_blk_size(num_blocks, output_blk_size, output.shape, axis);
  off_t offset = 0;

  for (GenericTensorAccessorR const &input : inputs) {
    size_t input_num_blocks = 1, input_blk_size = 1;
    calc_blk_size(input_num_blocks, input_blk_size, input.shape, axis);
    assert(input_num_blocks == num_blocks || output_blk_size == input_blk_size);

    int blocks_to_copy =
        (output_blk_size == input_blk_size) ? input_num_blocks : num_blocks;

    copy_with_stride<<<GET_BLOCKS(input_blk_size * num_blocks),
                       CUDA_NUM_THREADS,
                       0,
                       stream>>>(output.get_float_ptr() + offset,
                                 input.get_float_ptr(),
                                 blocks_to_copy,
                                 output_blk_size,
                                 input_blk_size);

    offset += (output_blk_size == input_blk_size)
                  ? input_blk_size * input_num_blocks
                  : input_blk_size;
  }
}

void backward_kernel(hipStream_t stream,
                     GenericTensorAccessorR const &output_grad,
                     std::vector<GenericTensorAccessorW> const &input_grads,
                     ff_dim_t axis) {
  assert(input_grads.size() <= MAX_NUM_INPUTS);
  size_t num_blocks = 1, output_blk_size = 1;
  calc_blk_size(num_blocks, output_blk_size, output_grad.shape, axis);
  off_t offset = 0;

  for (auto &input_grad : input_grads) {
    size_t input_num_blocks = 1, input_blk_size = 1;
    calc_blk_size(input_num_blocks, input_blk_size, input_grad.shape, axis);
    assert(input_num_blocks == num_blocks || output_blk_size == input_blk_size);

    int blocks_to_add =
        (output_blk_size == input_blk_size) ? input_num_blocks : num_blocks;

    add_with_stride<<<GET_BLOCKS(input_blk_size * num_blocks),
                      CUDA_NUM_THREADS,
                      0,
                      stream>>>(input_grad.get_float_ptr(),
                                output_grad.get_float_ptr() + offset,
                                blocks_to_add,
                                input_blk_size,
                                output_blk_size);

    offset += (output_blk_size == input_blk_size)
                  ? input_blk_size * input_num_blocks
                  : input_blk_size;
  }
}

} // namespace FlexFlow::Kernels::Concat
