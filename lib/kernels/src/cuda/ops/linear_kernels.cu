#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "internal/device.h"
#include "kernels/allocation.h"
#include "kernels/linear_kernels_gpu.h"
#include "utils/integer_conversions.h"

namespace FlexFlow {

namespace Kernels {
namespace Linear {

static bool use_activation(std::optional<Activation> activation) {
  if (activation.has_value()) {
    switch (activation.value()) {
      case Activation::RELU:
      case Activation::SIGMOID:
      case Activation::TANH:
        return true;
      case Activation::GELU:
        return false;
      default:
        assert(false && "Unsupported activation for Linear");
        break;
    }
  }
  return false;
}

LinearPerDeviceState
    gpu_init_kernel(PerDeviceFFHandle handle,
                    std::optional<Activation> activation,
                    std::optional<RegularizerAttrs> regularizer,
                    bool use_bias,
                    DataType input_type,
                    DataType weight_type,
                    DataType output_type,
                    int batch_size,
                    int channel) {
  ffTensorDescriptor_t outputTensor;
  ffActivationDescriptor_t actiDesc;
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor,
                                        HIPDNN_TENSOR_NCHW,
                                        ff_to_cudnn_datatype(output_type),
                                        batch_size,
                                        channel,
                                        1,
                                        1));
  hipdnnActivationMode_t mode;
  if (activation.has_value()) {
    switch (activation.value()) {
      case Activation::RELU:
        mode = HIPDNN_ACTIVATION_RELU;
        break;
      case Activation::SIGMOID:
        mode = HIPDNN_ACTIVATION_SIGMOID;
        break;
      case Activation::TANH:
        mode = HIPDNN_ACTIVATION_TANH;
        break;
      case Activation::GELU:
        // mode = CUDNN_ACTIVATION_GELU; //hipdnnActivationMode_t does not have
        // GELU
        break;
      default:
        // Unsupported activation mode
        assert(false);
    }
  } else {
    mode = HIPDNN_ACTIVATION_PATHTRU;
  }
  checkCUDNN(
      hipdnnSetActivationDescriptor(actiDesc, mode, HIPDNN_PROPAGATE_NAN, 0.0));
  // don't need this line below because we are already setting 4dDescriptor for
  // outputTensor above checkCUDNN(
  //     cudnnSetTensorDescriptorFromArrayShape(outputTensor, output_shape));

  // todo: how to use allocator to allocate memory for float * one_ptr, how many
  // bytes to allocate?
  float *one_ptr;
  checkCUDA(hipMalloc(&one_ptr, sizeof(float) * batch_size));
  float one_ptr_cpu[batch_size];
  for (int i = 0; i < batch_size; i++) {
    one_ptr_cpu[i] = 1.0;
  }
  checkCUDA(hipMemcpy(one_ptr,
                       one_ptr_cpu,
                       sizeof(float) * batch_size,
                       hipMemcpyHostToDevice));
  LinearPerDeviceState per_device_state = LinearPerDeviceState{
      /*handle=*/handle,
      /*outputTensor=*/outputTensor,
      /*actiDesc=*/actiDesc,
      /*one_ptr=*/one_ptr,
      /*mode=*/mode,
      /*activation=*/activation,
      /*regularizer=*/regularizer,
      /*use_bias=*/use_bias,
      /*input_type=*/input_type,
      /*weight_type=*/weight_type,
      /*output_type=*/output_type,
  };
  return per_device_state;
}

void gpu_forward_kernel(hipStream_t stream,
                        LinearPerDeviceState const &m,
                        float const *input_ptr,
                        float *output_ptr,
                        float const *weight_ptr,
                        float const *bias_ptr,
                        int in_dim,
                        int out_dim,
                        int batch_size) {

  checkCUBLAS(hipblasSetStream(m.handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m.handle.dnn, stream));
  float alpha = 1.0f, beta = 0.0f;
  hipDataType input_type = ff_to_cuda_datatype(m.input_type);
  hipDataType weight_type = ff_to_cuda_datatype(m.weight_type);
  hipDataType output_type = ff_to_cuda_datatype(m.output_type);
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = HIP_R_32F;
#endif
  checkCUBLAS(hipblasGemmEx(m.handle.blas,
                           HIPBLAS_OP_T,
                           HIPBLAS_OP_N,
                           out_dim,
                           batch_size,
                           in_dim,
                           &alpha,
                           static_cast<void const *>(weight_ptr),
                           weight_type,
                           in_dim,
                           static_cast<void const *>(input_ptr),
                           input_type,
                           in_dim,
                           &beta,
                           static_cast<void *>(output_ptr),
                           output_type,
                           out_dim,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  if (bias_ptr != nullptr) {
    checkCUBLAS(hipblasGemmEx(m.handle.blas,
                             HIPBLAS_OP_N,
                             HIPBLAS_OP_N,
                             out_dim,
                             batch_size,
                             1,
                             &alpha,
                             static_cast<void const *>(bias_ptr),
                             weight_type,
                             out_dim,
                             static_cast<void const *>(m.one_ptr),
                             HIP_R_32F,
                             1,
                             &alpha,
                             static_cast<void *>(output_ptr),
                             output_type,
                             out_dim,
                             compute_type,
                             CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  }
  // if (use_activation(m.activation)) {
  //   checkCUDNN(hipdnnActivationForward(m.handle.dnn,
  //                                     m.actiDesc,
  //                                     &alpha,
  //                                     m.outputTensor,
  //                                     static_cast<void *>(output_ptr),
  //                                     &beta,
  //                                     m.outputTensor,
  //                                     static_cast<void *>(output_ptr)));
  // } else if (m.activation == Activation::GELU) {
  //   size_t elements = size_t_from_int(out_dim) * size_t_from_int(batch_size);
  //   constexpr float B = 0.7978845608028654f;   // sqrt(2.0/M_PI)
  //   constexpr float C = 0.035677408136300125f; // 0.044715 * sqrt(2.0/M_PI)
  //   gelu_forward_kernel<<<GET_BLOCKS(elements), CUDA_NUM_THREADS>>>(
  //       elements, B, C, (float *)output_ptr);
  // } else {
  //   // Do nothing
  // }
}

void gpu_backward_kernel(hipStream_t stream,
                         LinearPerDeviceState const &m,
                         float const *output_ptr,
                         float *output_grad_ptr,
                         float const *input_ptr,
                         float *input_grad_ptr,
                         float const *kernel_ptr,
                         float *kernel_grad_ptr,
                         float *bias_grad_ptr,
                         int in_dim,
                         int out_dim,
                         int batch_size) {
  checkCUBLAS(hipblasSetStream(m.handle.blas, stream));
  checkCUDNN(hipdnnSetStream(m.handle.dnn, stream));
  float alpha = 1.0f;
  hipDataType input_type = ff_to_cuda_datatype(m.input_type);
  hipDataType weight_type = ff_to_cuda_datatype(m.weight_type);
  hipDataType output_type = ff_to_cuda_datatype(m.output_type);
#if CUDA_VERSION >= 11000
  // TODO: currently set the default to HIPBLAS_COMPUTE_16F for best performance
  hipblasComputeType_t compute_type = HIPBLAS_COMPUTE_16F;
#else
  hipDataType compute_type = HIP_R_32F;
#endif
  int output_size = out_dim * batch_size;
  if (m.activation.has_value()) {
    if (m.activation == Activation::RELU) {
      relu_backward_kernel(m.output_type,
                           static_cast<void *>(output_grad_ptr),
                           static_cast<void const *>(output_ptr),
                           output_size,
                           stream);
    } else if (m.activation == Activation::SIGMOID) {
      sigmoid_backward_kernel(m.output_type,
                              static_cast<void *>(output_grad_ptr),
                              static_cast<void const *>(output_ptr),
                              output_size,
                              stream);
    } else {
      // TODO: only support relu and sigmoid for now
      PANIC("Unsupported activation for Linear", m.activation.value());
    }
  }

  // Compute weight gradiant
  // NOTE: we use alpha=1 for kernel_grad to accumulate gradients
  checkCUBLAS(hipblasGemmEx(m.handle.blas,
                           HIPBLAS_OP_N,
                           HIPBLAS_OP_T,
                           in_dim,
                           out_dim,
                           batch_size,
                           &alpha,
                           static_cast<void const *>(input_ptr),
                           input_type,
                           in_dim,
                           static_cast<void *>(output_grad_ptr),
                           output_type,
                           out_dim,
                           &alpha,
                           static_cast<void *>(kernel_grad_ptr),
                           weight_type,
                           in_dim,
                           compute_type,
                           CUBLAS_GEMM_DEFAULT_TENSOR_OP));

  if (m.regularizer == std::nullopt) {
    // do nothing
  } else {
    RegularizerAttrs regularizer_attrs = m.regularizer.value();
    if (regularizer_attrs.has<L2RegularizerAttrs>()) {
      L2RegularizerAttrs l2_attrs = regularizer_attrs.get<L2RegularizerAttrs>();
      float lambda = l2_attrs.lambda;
      checkCUBLAS(hipblasSgeam(m.handle.blas,
                              HIPBLAS_OP_N,
                              HIPBLAS_OP_N,
                              in_dim,
                              out_dim,
                              &alpha,
                              kernel_grad_ptr,
                              in_dim,
                              &lambda,
                              kernel_ptr,
                              in_dim,
                              kernel_grad_ptr,
                              in_dim));
    } else {
      assert(false && "Only L2 regularization is supported");
    }
  }

  // Compute bias gradiant
  // NOTE: we use alpha=1 for bias_grad to accumulate gradients
  // use_bias = True
  if (bias_grad_ptr != NULL) {
    checkCUBLAS(hipblasGemmEx(m.handle.blas,
                             HIPBLAS_OP_N,
                             HIPBLAS_OP_T,
                             1,
                             out_dim,
                             batch_size,
                             &alpha,
                             static_cast<void const *>(m.one_ptr),
                             HIP_R_32F,
                             1,
                             static_cast<void *>(output_grad_ptr),
                             output_type,
                             out_dim,
                             &alpha,
                             static_cast<void *>(bias_grad_ptr),
                             weight_type,
                             1,
                             compute_type,
                             CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  }
  // Compute data gradiant
  // NOTE: we use alpha=1 for input_grad to accumulate gradients
  if (input_grad_ptr != NULL) {
    checkCUBLAS(hipblasGemmEx(m.handle.blas,
                             HIPBLAS_OP_N,
                             HIPBLAS_OP_N,
                             in_dim,
                             batch_size,
                             out_dim,
                             &alpha,
                             static_cast<void const *>(kernel_ptr),
                             weight_type,
                             in_dim,
                             static_cast<void *>(output_grad_ptr),
                             output_type,
                             out_dim,
                             &alpha,
                             static_cast<void *>(input_grad_ptr),
                             input_type,
                             in_dim,
                             compute_type,
                             CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  }
}

void gpu_cleanup_kernel(LinearPerDeviceState &per_device_state) {
  NOT_IMPLEMENTED();
}

} // namespace Linear
} // namespace Kernels
} // namespace FlexFlow
