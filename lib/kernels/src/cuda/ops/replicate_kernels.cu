#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "internal/device.h"
#include "kernels/datatype_dispatch.h"
#include "kernels/replicate_kernels_gpu.h"

namespace FlexFlow {
namespace Kernels {
namespace Replicate {

template <typename T>
__global__ void replicate_backward_kernel(T const *output_ptr,
                                          T *input_ptr,
                                          size_t num_elements,
                                          size_t num_replicas) {
  CUDA_KERNEL_LOOP(i, num_elements) {
    for (size_t j = 0; j < num_replicas; j++) {
      input_ptr[i] += output_ptr[i + j * num_elements];
    }
  }
}

template <DataType T>
struct ForwardKernel {
  void operator()(hipStream_t stream,
                  GenericTensorAccessorR const &input,
                  GenericTensorAccessorW const &output) {
    checkCUDA(
        hipMemcpyAsync((void *)output.get<T>(),
                        (void *)input.get<T>(),
                        input.shape.num_elements().int_from_positive_int() *
                            size_of_datatype(T).int_from_positive_int(),
                        hipMemcpyDeviceToDevice,
                        stream));
  }
};

template <DataType T>
struct BackwardKernel {
  void operator()(hipStream_t stream,
                  GenericTensorAccessorR const &output,
                  GenericTensorAccessorW const &input,
                  size_t num_replicas) {
    size_t total_elements =
        input.shape.num_elements().int_from_positive_int() * num_replicas;
    replicate_backward_kernel<real_type_t<T>>
        <<<GET_BLOCKS(total_elements), CUDA_NUM_THREADS, 0, stream>>>(
            output.get<T>(),
            input.get<T>(),
            input.shape.num_elements().int_from_positive_int(),
            num_replicas);
  }
};

void gpu_forward_kernel(hipStream_t stream,
                    GenericTensorAccessorR const &input,
                    GenericTensorAccessorW const &output) {
  DataTypeDispatch1<ForwardKernel>{}(input.data_type, stream, input, output);
}

void gpu_backward_kernel(hipStream_t stream,
                     GenericTensorAccessorR const &output,
                     GenericTensorAccessorW const &input,
                     size_t num_replicas) {
  DataTypeDispatch1<BackwardKernel>{}(
      input.data_type, stream, output, input, num_replicas);
}

} // namespace Replicate
} // namespace Kernels
} // namespace FlexFlow
