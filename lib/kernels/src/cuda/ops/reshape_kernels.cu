#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "internal/device.h"
#include "kernels/datatype_dispatch.h"
#include "kernels/reshape_kernels_gpu.h"

namespace FlexFlow {

namespace Kernels {
namespace Reshape {

template <DataType InputDT, DataType OutputDT>
struct ForwardKernel {
  void operator()(hipStream_t stream,
                  GenericTensorAccessorR const &input,
                  GenericTensorAccessorW const &output) {
    checkCUDA(
        hipMemcpyAsync(output.get<OutputDT>(),
                        input.get<InputDT>(),
                        input.shape.num_elements().int_from_positive_int() *
                            size_of_datatype(OutputDT).int_from_positive_int(),
                        hipMemcpyDeviceToDevice,
                        stream));
  }
};

template <typename DT, typename DTGrad>
__global__ void apply_add_with_scale2(DT *data_ptr,
                                      DTGrad const *grad_ptr,
                                      size_t size,
                                      DT scale) {
  CUDA_KERNEL_LOOP(i, size) {
    data_ptr[i] += grad_ptr[i] * scale;
  }
}

template <DataType InputDT, DataType OutputDT>
struct BackwardKernel {
  void operator()(hipStream_t stream,
                  GenericTensorAccessorR const &output,
                  GenericTensorAccessorW const &input) {
    float alpha = 1.0f;
    apply_add_with_scale2<real_type_t<InputDT>, real_type_t<OutputDT>>
        <<<GET_BLOCKS(input.shape.num_elements().int_from_positive_int()),
           CUDA_NUM_THREADS,
           0,
           stream>>>(input.get<InputDT>(),
                     output.get<OutputDT>(),
                     input.shape.num_elements().int_from_positive_int(),
                     static_cast<real_type_t<InputDT>>(alpha));
  }
};

void gpu_forward_kernel(hipStream_t stream,
                        GenericTensorAccessorR const &input,
                        GenericTensorAccessorW const &output) {
  DataTypeDispatch2<ForwardKernel>{}(
      input.data_type, output.data_type, stream, input, output);
}

void gpu_backward_kernel(hipStream_t stream,
                         GenericTensorAccessorR const &output,
                         GenericTensorAccessorW const &input) {
  DataTypeDispatch2<BackwardKernel>{}(
      input.data_type, output.data_type, stream, output, input);
}

} // namespace Reshape
} // namespace Kernels
} // namespace FlexFlow
